
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<string.h>
#include<new>

#define FALSE 0
#define TRUE 1
#define STR_EQ 0

#define max(a, b) \
	({__typeof__ (a) _a = (a); \
		__typeof__ (b) _b = (b); \
		_a > _b ? _a : _b; })
		
#define min(a, b) \
	({__typeof__ (a) _a = (a); \
		__typeof__ (b) _b = (b); \
		_a > _b ? _b : _a; })
		
#define abs(a) \
	({__typeof__ (a) _a = (a); \
		_a >= 0 ? _a : -_a; })

/* =================== BASIC FUNCTIONS =====================================================================*/
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ void curandInit(hiprandState_t* state_ptr, int tid){
	hiprand_init((unsigned long long)clock(), tid, 0, state_ptr);
}

__device__ float cudaFloatRand(float min, float max, hiprandState_t* state_ptr){
	return min + hiprand_uniform(state_ptr) * (max - min);
}

__device__ int cudaIntRand(int min, int max, hiprandState_t* state_ptr){
	return int(cudaFloatRand(float(min), float(max + 1.0), state_ptr));
}

__host__ float floatRand(float min, float max){
	float scale = rand() / (float) RAND_MAX;
	return min + scale * (max - min);
}	

__host__ char roll(float probability){
	if(floatRand(0.0, 1.0) < probability)
		return TRUE;
	return FALSE;
}	
__device__ char cudaRoll(float probability, hiprandState_t* curand_state_ptr){
	if(cudaFloatRand(0.0, 1.0, curand_state_ptr) < probability)
		return TRUE;
	return FALSE;
}

/* =================== STRUCTS AND METHODS =====================================================================*/

typedef struct SimulationOptions{
	int N;
	float DIM;
	int simulation_time;
	float infection_r;
	float infection_p;
	float velocity;
	int immune_time;
	int sympthoms_time;
	int blocks;
	int threads_per_block;
	char* output;
	float lawful_p;
	int quarantine_sick_time;
	int quarantine_all_time;
	int gathering_points_n;
	float gathering_point_p;
	int buffor_size;
} SimulationOptions;

typedef enum{HEALTHY, CARRIER, SICK, IMMUNE} Health;
typedef enum{GOING_TO, GOING_BACK, NO_DESTINATION} GatheringPointTravel;

typedef struct Point{
	float x;
	float y;
} Point;

__host__ Point randPoint(float DIM){
	Point point;
	point.x = floatRand(0.0, DIM);
	point.y = floatRand(0.0, DIM);
	return point;
}

__device__ Point cudaRandPoint(float DIM, hiprandState_t* state_ptr){
	Point point;
	point.x = cudaFloatRand(0.0, DIM, state_ptr);
	point.y = cudaFloatRand(0.0, DIM, state_ptr);
	return point;
}

__host__ __device__ float distance(Point p1, Point p2){
	float dx = abs(p1.x - p2.x);
	float dy = abs(p1.y - p2.y);
	return sqrt(dx * dx + dy * dy);
}

typedef struct Person{
	Point location;
	Point home;
	Health health;
	GatheringPointTravel travel;
	char quarantined; // SICK people are totaly quarantined, the rest is partialy quarantined
	int time_sick;
	Point destination;
	char lawful;
} Person;

typedef struct PersonInfo{
	Point location;
	Health health;
} PersonInfo;

/* =================== DEVICE CODE =====================================================================*/

__device__ void updateQuarantine(SimulationOptions settings, Person* person_ptr, int time){
	if(!(person_ptr->lawful))
		return;
	if(settings.quarantine_all_time && settings.quarantine_all_time < time)
		person_ptr->quarantined = TRUE;
	else if(settings.quarantine_sick_time && settings.quarantine_sick_time < time){
		if(person_ptr->health == SICK){
			person_ptr->quarantined = TRUE;
			person_ptr->travel = NO_DESTINATION;
		}
		else
			person_ptr->quarantined = FALSE;
	}
}

__device__ void migrate(
	SimulationOptions settings,
	Person* person_ptr,
	hiprandState_t* state_ptr,
	Point* gathering_points
){
	float angle, dy, dx;
	float destination_r = settings.velocity;
	
	if(person_ptr->quarantined){
		if(person_ptr->health == SICK)
			return;
		if(person_ptr->travel == GOING_TO && distance(person_ptr->location, person_ptr->destination) < destination_r){
			person_ptr->destination = person_ptr->home;
			person_ptr->travel = GOING_BACK;
		}
		if(person_ptr->travel == GOING_BACK && distance(person_ptr->location, person_ptr->destination) < destination_r){
			person_ptr->travel = NO_DESTINATION;
		}
		if(person_ptr->travel == NO_DESTINATION){
			if(!settings.gathering_points_n)
				return;
			if(!cudaRoll(settings.gathering_point_p, state_ptr))
				return;
			person_ptr->destination = gathering_points[cudaIntRand(0, settings.gathering_points_n - 1, state_ptr)];
			person_ptr->travel = GOING_TO;
		}
	}
	
	else if(distance(person_ptr->location, person_ptr->destination) < destination_r){
		person_ptr->destination = cudaRandPoint(settings.DIM, state_ptr);
	}
	
	dy = person_ptr->destination.y - person_ptr->location.y;
	dx = person_ptr->destination.x - person_ptr->location.x;
	angle = atan2(dy, dx);
	person_ptr->location.x = min(max(person_ptr->location.x + cos(angle) * settings.velocity, 0.0), settings.DIM);
	person_ptr->location.y = min(max(person_ptr->location.y + sin(angle) * settings.velocity, 0.0), settings.DIM);
}

__device__ void developDisease(SimulationOptions settings, Person* person_ptr){
	if(person_ptr->health == CARRIER || person_ptr->health == SICK)
		person_ptr->time_sick += 1;
	if(person_ptr->time_sick > settings.immune_time)
		person_ptr->health = IMMUNE;
	else if(person_ptr->time_sick > settings.sympthoms_time)
		person_ptr->health = SICK;
}

// there may be races, but it doesn't matter (I think?)
__device__ void infect(
	SimulationOptions settings,
	Person* population,
	int me_idx,
	hiprandState_t* curand_state_ptr
){
	Person* me_ptr = &population[me_idx];
	Person* person_ptr;
	int i;
	if((me_ptr->health == CARRIER || me_ptr->health == SICK) && !(me_ptr->quarantined && me_ptr->health == SICK)){
		for(i = 0; i < settings.N; i++){
			person_ptr = &population[i];
			if(i == me_idx) continue;
			if(person_ptr->quarantined && person_ptr->travel == NO_DESTINATION) continue;
			if(person_ptr->health == CARRIER || person_ptr->health == SICK) continue;
			if(distance(me_ptr->location, person_ptr->location) > settings.infection_r) continue;
			if(cudaRoll(settings.infection_p, curand_state_ptr))
				person_ptr->health = CARRIER;
		}
	}
}

__global__ void simulate(
	SimulationOptions settings,
	Person* population,
	hiprandState_t* curand_states,
	int time,
	Point* gathering_points,
	int buffor_index,
	PersonInfo* population_info
){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	Person* person_ptr;
	hiprandState_t my_curand_state = curand_states[tid];
	curandInit(&my_curand_state, tid);
	
	// develop disease
	i = tid;
	while(i < settings.N){
		person_ptr = &population[i];
		developDisease(settings, person_ptr);
		i += gridDim.x * blockDim.x;
	}
	
	// update population quarantine_all_time
	i = tid;
	while(i < settings.N){
		person_ptr = &population[i];
		updateQuarantine(settings, person_ptr, time);
		i += gridDim.x * blockDim.x;
	}
	
	// migration of population
	i = tid;
	while(i < settings.N){
		person_ptr = &population[i];
		migrate(settings, person_ptr, &my_curand_state, gathering_points);
		i += gridDim.x * blockDim.x;
	}
	
	// spread of disease
	i = tid;
	while(i < settings.N){
		infect(settings, population, i, &my_curand_state);
		i += gridDim.x * blockDim.x;
	}
	
	// save to buffor
	i = tid;
	while(i < settings.N){
		population_info[settings.N * buffor_index + i].location = population[i].location;
		population_info[settings.N * buffor_index + i].health = population[i].health;
		i += gridDim.x * blockDim.x;
	}
}

/* =================== HOST =====================================================================*/

int main(int argc, char** argv){
	SimulationOptions settings;
	int i, j, buffors_simulated;
	FILE* file;
	char save_output;
	Person* population;
	Person* dev_population;
	hiprandState_t* curand_states;
	Point* gathering_points;
	Point* dev_gathering_points;
	PersonInfo* population_info;
	PersonInfo* dev_population_info;
	
	settings.N = 10000;
	settings.DIM = 100;
	settings.simulation_time = 500;
	settings.velocity = 1.0;
	settings.infection_p = 0.33;
	settings.infection_r = 3.0;
	settings.immune_time = 100;
	settings.sympthoms_time = 10;
	settings.blocks = 128;
	settings.threads_per_block = 128;
	settings.output = "output.sim";
	settings.quarantine_all_time = 0;
	settings.quarantine_sick_time = 0;
	settings.lawful_p = 1.0;
	settings.gathering_points_n = 0;
	settings.gathering_point_p = 0.05;
	settings.buffor_size = 1;
	
	//read commandline args
	i = 1;
	while(i < argc - 1){
		if(strcmp(argv[i], "--N") == STR_EQ || strcmp(argv[i], "-N") == STR_EQ){
			settings.N = atoi(argv[++i]);
			if(settings.N < 1) return 1;
		}
		else if(strcmp(argv[i], "-=DIM") == STR_EQ || strcmp(argv[i], "-DIM") == STR_EQ){
			settings.DIM = atof(argv[++i]);
			if(settings.DIM <= 0.0) return 1;
		}
		else if(strcmp(argv[i], "--simulation_n") == STR_EQ || strcmp(argv[i], "-simn") == STR_EQ){
			settings.simulation_time = atoi(argv[++i]);
			if(settings.simulation_time < 1) return 1;
		}
		else if(strcmp(argv[i], "--velocity") == STR_EQ || strcmp(argv[i], "-v") == STR_EQ){
			settings.velocity = atof(argv[++i]);
			if(settings.velocity < 0) return 1;
		}
		else if(strcmp(argv[i], "--infection_p") == STR_EQ || strcmp(argv[i], "-infp") == STR_EQ){
			settings.infection_p = atof(argv[++i]);
			if(settings.infection_p <= 0.0) return 1;
		}
		else if(strcmp(argv[i], "--infection_r") == STR_EQ || strcmp(argv[i], "-infr") == STR_EQ){
			settings.infection_r = atof(argv[++i]);
			if(settings.infection_r <= 0.0) return 1;
		}
		else if(strcmp(argv[i], "--immune_time") == STR_EQ || strcmp(argv[i], "-immt") == STR_EQ){
			settings.immune_time = atoi(argv[++i]);
			if(settings.immune_time < 0) return 1;
		}
		else if(strcmp(argv[i], "--sympthoms_time") == STR_EQ || strcmp(argv[i], "-symt") == STR_EQ){
			settings.sympthoms_time = atoi(argv[++i]);
			if(settings.sympthoms_time < 0) return 1;
		}
		else if(strcmp(argv[i], "--blocks") == STR_EQ || strcmp(argv[i], "-b") == STR_EQ){
			settings.blocks = atoi(argv[++i]);
			if(settings.blocks < 1) return 1;
		}
		else if(strcmp(argv[i], "--threads_per_block") == STR_EQ || strcmp(argv[i], "-tpb") == STR_EQ){
			settings.threads_per_block = atoi(argv[++i]);
			if(settings.threads_per_block < 1) return 1;
		}
		else if(strcmp(argv[i], "--output") == STR_EQ || strcmp(argv[i], "-o") == STR_EQ){
			settings.output = argv[++i];
			if(!settings.output) return 1;
		}
		else if(strcmp(argv[i], "--quarantine_all_time") == STR_EQ || strcmp(argv[i], "-qat") == STR_EQ){
			settings.quarantine_all_time = atoi(argv[++i]);
			if(settings.quarantine_all_time < 0) return 1;
		}
		else if(strcmp(argv[i], "--quarantine_sick_time") == STR_EQ || strcmp(argv[i], "-qst") == STR_EQ){
			settings.quarantine_sick_time = atoi(argv[++i]);
			if(settings.quarantine_sick_time < 0) return 1;
		}
		else if(strcmp(argv[i], "--lawful_p") == STR_EQ || strcmp(argv[i], "-lawp") == STR_EQ){
			settings.lawful_p = atof(argv[++i]);
			if(settings.lawful_p < 0.0) return 1;
		}
		else if(strcmp(argv[i], "--gathering_points_n") == STR_EQ || strcmp(argv[i], "-gn") == STR_EQ){
			settings.gathering_points_n = atoi(argv[++i]);
			if(settings.gathering_points_n < 0) return 1;
		}
		else if(strcmp(argv[i], "--gathering_point_p") == STR_EQ || strcmp(argv[i], "-gp") == STR_EQ){
			settings.gathering_point_p = atof(argv[++i]);
			if(settings.gathering_point_p < 0.0) return 1;
		}
		else if(strcmp(argv[i], "--buffor_size") == STR_EQ || strcmp(argv[i], "-buff") == STR_EQ){
			settings.buffor_size = atoi(argv[++i]);
			if(settings.buffor_size < 1) return 1;
		}
		i++;
	}
	
	
	
	if(strcmp(settings.output, "none") == STR_EQ)
		save_output = FALSE;
	else
		save_output = TRUE;
	
	try{
		population_info = new PersonInfo[settings.N * settings.buffor_size];
		population = new Person[settings.N];
	}
	catch(const std::bad_alloc& e){
		printf("Insufficent memory on host\n");
		return 1;
	}
	
	
	srand((unsigned int)time(NULL));
	
	for(i = 0; i < settings.N; i++){
		population[i].location.x = floatRand(0.0, settings.DIM);
		population[i].location.y = floatRand(0.0, settings.DIM);
		population[i].home = population[i].location;
		population[i].destination.x = floatRand(0.0, settings.DIM);
		population[i].destination.y = floatRand(0.0, settings.DIM);
		population[i].health = HEALTHY;
		population[i].quarantined = FALSE;
		population[i].time_sick = 0;
		population[i].travel = NO_DESTINATION;
		if(roll(settings.lawful_p))
			population[i].lawful = TRUE;
		else
			population[i].lawful = FALSE;
	}
	
	gathering_points = new Point[settings.gathering_points_n];
	for(i = 0; i < settings.gathering_points_n; i++){
		gathering_points[i].x = floatRand(0.0, settings.DIM);
		gathering_points[i].y = floatRand(0.0, settings.DIM);
	}
	
	
	//patient zero
	population[0].health = CARRIER;
	
	HANDLE_ERROR( hipMalloc((void**)&dev_population, sizeof(Person) * settings.N) );
	HANDLE_ERROR( hipMalloc((void**)&curand_states, sizeof(hiprandState_t) * settings.blocks * settings.threads_per_block) );
	HANDLE_ERROR( hipMalloc((void**)&dev_gathering_points, sizeof(Point) * settings.gathering_points_n) );
	HANDLE_ERROR( hipMalloc((void**)&dev_population_info, sizeof(PersonInfo) * settings.N * settings.buffor_size) );
	
	HANDLE_ERROR( hipMemcpy(dev_population, population, sizeof(Person) * settings.N, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(dev_gathering_points, gathering_points, sizeof(Point) * settings.gathering_points_n, hipMemcpyHostToDevice) );

	if(save_output){
		file = fopen(settings.output, "w");
		fprintf(file, "%d %f %d %d\n", settings.N, settings.DIM, settings.simulation_time, settings.gathering_points_n);
		for(i = 0; i < settings.gathering_points_n; i++)
			fprintf(file, "%f %f\n", gathering_points[i].x, gathering_points[i].y);
	}
	// for(i = 0; i < settings.simulation_time; i++){
		// printf("==========SIM%d==========\n", i);
		// simulate<<<settings.blocks, settings.threads_per_block>>>(settings, dev_population, curand_states, i, dev_gathering_points);
		// cudaDeviceSynchronize();
		// HANDLE_ERROR( cudaMemcpy(population, dev_population, sizeof(Person) * settings.N, cudaMemcpyDeviceToHost) );
		// if(save_output){
			// for(j = 0; j < settings.N; j++){
				// fprintf(file, "%f %f %d\n", population[j].location.x, population[j].location.y, population[j].health);
			// }
		// }
	// }
	i = 0;
	while(i < settings.simulation_time){
		for(j = 0; j < settings.buffor_size; j++){
			printf("==========SIM%d==========\n", i);
			simulate<<<settings.blocks, settings.threads_per_block>>>(
				settings, dev_population, curand_states, i, dev_gathering_points, j, dev_population_info
			);
			hipDeviceSynchronize();
			buffors_simulated = j + 1;
			i++;
			if(i >= settings.simulation_time)
				break;
		}
		printf("Coping buffor from GPU to host...\n");
		HANDLE_ERROR( hipMemcpy(
			population_info, dev_population_info, sizeof(PersonInfo) * settings.N * settings.buffor_size, hipMemcpyDeviceToHost
		) );
		if(save_output){
			for(j = 0; j < settings.N * buffors_simulated; j++){
				fprintf(file, "%f %f %d\n", population_info[j].location.x, population_info[j].location.y, population_info[j].health);
			}
		}
	}
	
	if(save_output)
		fclose(file);
	hipFree(curand_states);
	hipFree(dev_population);
	hipFree(dev_gathering_points);
	hipFree(dev_population_info);
	delete[] population;
	delete[] gathering_points;
	delete[] population_info;
	return 0;
}